
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_say_hello(){
    printf("Hello world, CUDA! %d\n", threadIdx.x);
}

int main(){
    printf("Hello world, CPU\n");
    cuda_say_hello<<<1,1>>>();

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    return 0;
}
